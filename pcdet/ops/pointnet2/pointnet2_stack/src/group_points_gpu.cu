#include "hip/hip_runtime.h"
/*
Stacked-batch-data version of point grouping, modified from the original implementation of official PointNet++ codes.
Written by Shaoshuai Shi
All Rights Reserved 2019-2020.
*/


#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"
#include "group_points_gpu.h"


__global__ void group_points_grad_kernel_stack(int B, int M, int C, int N, int nsample,
    const float *grad_out, const int *idx, const int *idx_batch_cnt, const int *features_batch_cnt, float *grad_features) {
    // :param grad_out: (M1 + M2 ..., C, nsample) tensor of the gradients of the output from forward
    // :param idx: (M1 + M2 ..., nsample) tensor containing the indicies of features to group with
    // :param idx_batch_cnt: (batch_size) [M1 + M2 ...] tensor containing the indicies of features to group with
    // :param features_batch_cnt: (batch_size) [N1 + N2 ...] tensor containing the indicies of features to group with
    // :return:
    //     grad_features: (N1 + N2 ..., C) gradient of the features
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int sample_idx = index % nsample;
    int C_idx = (index / nsample) % C;
    int pt_idx = (index / nsample / C);

    if (pt_idx >= M || C_idx >= C || sample_idx >= nsample) return;

    int bs_idx = 0, pt_cnt = idx_batch_cnt[0];
    for (int k = 1; k < B; k++){
        if (pt_idx < pt_cnt) break;
        pt_cnt += idx_batch_cnt[k];
        bs_idx = k;
    }

    int features_batch_start_idx = 0;
    for (int k = 0; k < bs_idx; k++) features_batch_start_idx += features_batch_cnt[k];

    grad_out += pt_idx * C * nsample + C_idx * nsample + sample_idx;
    idx += pt_idx * nsample + sample_idx;
    grad_features += (features_batch_start_idx + idx[0]) * C + C_idx;

    atomicAdd(grad_features, grad_out[0]);
}

void group_points_grad_kernel_launcher_stack(int B, int M, int C, int N, int nsample,
    const float *grad_out, const int *idx, const int *idx_batch_cnt, const int *features_batch_cnt, float *grad_features) {
    // :param grad_out: (M1 + M2 ..., C, nsample) tensor of the gradients of the output from forward
    // :param idx: (M1 + M2 ..., nsample) tensor containing the indicies of features to group with
    // :param idx_batch_cnt: (batch_size) [M1 + M2 ...] tensor containing the indicies of features to group with
    // :param features_batch_cnt: (batch_size) [N1 + N2 ...] tensor containing the indicies of features to group with
    // :return:
    //     grad_features: (N1 + N2 ..., C) gradient of the features

    hipError_t err;
    // dim3 blocks(DIVUP(npoints * nsample, THREADS_PER_BLOCK), c, b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 blocks(DIVUP(M * C * nsample, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    group_points_grad_kernel_stack<<<blocks, threads>>>(B, M, C, N, nsample, grad_out, idx, idx_batch_cnt, features_batch_cnt, grad_features);

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


__global__ void group_points_kernel_stack(int B, int M, int C, int nsample,
    const float *features, const int *features_batch_cnt, const int *idx, const int *idx_batch_cnt, float *out) {
    // M, nsample => M, C, nsample
    // :param features: (N1 + N2 ..., C) tensor of features to group, sparse tensor跟spconv一样
    // :param features_batch_cnt: (batch_size) [N1 + N2 ...] tensor containing the indicies of features to group with
    //                            用来标记batch idx
    // :param idx: (M1 + M2 ..., nsample) tensor containing the indicies of features to group with
    //             ballquery函数的返回值，用来给每个pcl点选择voxel_center点
    // :param idx_batch_cnt: (batch_size) [M1 + M2 ...] tensor containing the indicies of features to group with，标记batch
    // :return:
    //     output: (M1 + M2, C, nsample) tensor，把c长度feature附在query上。
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int sample_idx = index % nsample;
    // index / nsample
    int C_idx = (index / nsample) % C;  // 一个thread只操作一个channel
    int pt_idx = (index / nsample / C);  // 每个点有c*nsample个元素
    
    // 上限M个点，C个channel，nsample个neighbor
    if (pt_idx >= M || C_idx >= C || sample_idx >= nsample) return;
    
    // 当前点所在batch idx
    int bs_idx = 0, pt_cnt = idx_batch_cnt[0];
    for (int k = 1; k < B; k++){
        if (pt_idx < pt_cnt) break;
        pt_cnt += idx_batch_cnt[k];
        bs_idx = k;
    }
    
    // feature在当前batch idx第一个位置
    int features_batch_start_idx = 0;
    for (int k = 0; k < bs_idx; k++) features_batch_start_idx += features_batch_cnt[k];
    features += features_batch_start_idx * C;

    idx += pt_idx * nsample + sample_idx;
    // 最后一个维度最先增长。
    int in_idx = idx[0] * C + C_idx;
    int out_idx = pt_idx * C * nsample + C_idx * nsample + sample_idx;

    out[out_idx] = features[in_idx];
}


void group_points_kernel_launcher_stack(int B, int M, int C, int nsample,
    const float *features, const int *features_batch_cnt, const int *idx, const int *idx_batch_cnt, float *out) {
    // :param features: (N1 + N2 ..., C) tensor of features to group
    // :param features_batch_cnt: (batch_size) [N1 + N2 ...] tensor containing the indicies of features to group with
    // :param idx: (M1 + M2 ..., nsample) tensor containing the indicies of features to group with
    // :param idx_batch_cnt: (batch_size) [M1 + M2 ...] tensor containing the indicies of features to group with
    // :return:
    //     output: (M1 + M2, C, nsample) tensor

    hipError_t err;
    dim3 blocks(DIVUP(M * C * nsample, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    group_points_kernel_stack<<<blocks, threads>>>(B, M, C, nsample, features, features_batch_cnt, idx, idx_batch_cnt, out);
    // hipDeviceSynchronize();  // for using printf in kernel function
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
